#include "hip/hip_runtime.h"
#include <stdio.h>

extern "C" {
#include <header.h>
}

__global__ void
burning_ship_kernel(Uint32 *a, int rx, int ry)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int dim_i = y * rx + x;
	if ((x >= rx) || (y >= ry))
		return ;

	float GraphTop = 1.5f;
	float GraphBottom = -1.5f;
	float GraphLeft = -2.0f;
	float GraphRight = 1.5f;
	int i;
	int max_iteration = 256;

	float incrementX = ((GraphRight - GraphLeft) / (rx - 1));
	float DecrementY = ((GraphTop - GraphBottom) / (ry - 1));
	float Zx, Zy;
	float CoordReal;
	float CoordImaginary = GraphTop;
	float SquaredX, SquaredY;

	int palette[256];
	for (int n = 0; n < 256; n++)
	{
		palette[n] = (int)(n + 512 - 512 * expf(-n / 50.0) / 3.0);
		palette[n] = palette[n] << 24 | palette[n] << 16 | palette[n] << 8 | 255;
	}
	palette[255] = 0;

	CoordReal = GraphLeft + (incrementX * x);
	CoordImaginary = GraphTop - (DecrementY * y);
	i = 0;
	Zx = CoordReal;
	Zy = CoordImaginary;
	SquaredX = Zx * Zx;
	SquaredY = Zy * Zy;
	a[dim_i] = 0;
	while ((i < max_iteration) && ((SquaredX + SquaredY) < 4.0))
	{
		Zy = fabs(Zx * Zy);
		Zy = Zy + Zy - CoordImaginary;
		Zx = SquaredX - SquaredY + CoordReal;
		SquaredX = Zx * Zx;
		SquaredY = Zy * Zy;
		i++;
	}
	a[dim_i] = palette[(i + 1) % 255];
}

void
set_palette(int palette[256])
{
	for (int n = 0; n < 256; n++)
	{
		palette[n] = (int)(n + 512 - 512 * expf(-n / 50.0) / 3.0);
		palette[n] = palette[n] << 24 | palette[n] << 16 | palette[n] << 8 | 255;
	}
	palette[255] = 0;
}

extern "C" void
burning_ship(t_data *data)
{
	static Uint32 *a_d = NULL;  // Pointer to host & device arrays

	size_t size = SDL_RY * SDL_RX * data->surf->format->BytesPerPixel;

	if (a_d == NULL)
		hipMalloc((void **)&a_d, size);   // Allocate array on device

	dim3 blockSize(32, 32);
	int bx = (SDL_RX + blockSize.x - 1) / blockSize.x;
	int by = (SDL_RY + blockSize.y - 1) / blockSize.y;
	dim3 gridSize = dim3(bx, by);

	//int palette[256];
	//set_palette(palette);
	burning_ship_kernel<<<gridSize, blockSize>>>(a_d, SDL_RX, SDL_RY);
	hipDeviceSynchronize();

	SDL_LockSurface(data->surf);
	// Retrieve result from device and store it in host array
	hipMemcpy(data->surf->pixels, a_d, size, hipMemcpyDeviceToHost);
	SDL_UnlockSurface(data->surf);

	//cleanup
	if (data->esdl->run == 0)
		hipFree(a_d);
}