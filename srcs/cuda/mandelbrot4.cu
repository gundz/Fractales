#include "hip/hip_runtime.h"
extern "C"
{
#include <header.h>
}
#include <hip/hip_runtime.h>
#include "tools.cu"

__global__ void
mandelbrot4_kernel(t_cuda cuda, t_fractal fractal)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int dim_i = y * cuda.rx + x;
	if ((x >= cuda.rx) || (y >= cuda.ry))
		return ;

	double		pr, pi;
	double		zx, zy;
	double		zx2, zy2;
	int			i;

	pr = fractal.cx + (x - cuda.rx / 2) * fractal.zoom + fractal.movex;
	pi = fractal.cy + (y - cuda.ry / 2) * fractal.zoom + fractal.movey;
	zx = 0;
	zy = 0;
	i = 0;
	while (++i < fractal.maxiteration)
	{
		zy2 = zy * zy;
		zx2 = zx * zx;
		zx = (zx2 * zx2) - (6 * zx2 * zy2) + (zy2 * zy2) + pr;
		zy = (4 * (zx2 * zx) * zy) - (4 * zx * (zy2 * zy)) + pi;
		if (zx2 + zy2 >= 4)
			break ;
	}
	int brightness = cuda_color_it(zx2, zy2, i, 100);
	cuda.screen[dim_i] = hsv_to_rgb(brightness % 256, 255, 255 * (i < fractal.maxiteration));
}

int
mandelbrot4_call(t_data *data, t_cuda *cuda)
{
	static t_fractal fractal = {(2.5 / SDL_RY), 0, 0, 400, 0, 0, 0, 0, 0, 0};

	fractal.mx = data->esdl->en.in.m_x - SDL_RX / 2;
	fractal.my = data->esdl->en.in.m_y - SDL_RY / 2;
	fractal_input(data, &fractal);
	mandelbrot4_kernel<<<cuda->gridsize, cuda->blocksize>>>(*cuda, fractal);
	return (0);
}

void
mandelbrot4(t_data *data)
{
	do_cuda(data, &mandelbrot4_call);
}
