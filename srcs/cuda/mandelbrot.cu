#include "hip/hip_runtime.h"
extern "C"
{
#include <header.h>
}
#include <hip/hip_runtime.h>
#include <mandelbrot.h>

__global__ void
mandelbrot_kernel(t_cuda cuda, t_mandelbrot mandelbrot)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int dim_i = y * cuda.rx + x;
	if ((x >= cuda.rx) || (y >= cuda.ry))
		return ;

	//each iteration, it calculates: newz = oldz*oldz + p, where p is the current pixel, and oldz stars at the origin
	double pr, pi;           //real and imaginary part of the pixel p
	double newRe, newIm, oldRe, oldIm;   //real and imaginary parts of new and old z

	//calculate the initial real and imaginary part of z, based on the pixel location and zoom and position values
	pr = 1.5 * (x - cuda.rx / 2) / (0.5 * mandelbrot.zoom * cuda.rx) + mandelbrot.moveX;
	pi = (y - cuda.ry / 2) / (0.5 * mandelbrot.zoom * cuda.ry) + mandelbrot.moveY;
	newRe = newIm = oldRe = oldIm = 0; //these should start at 0,0

	int i;
	//start the iteration process
	for(i = 0; i < mandelbrot.maxIteration; i++)
	{
		//remember value of previous iterations
		oldRe = newRe;
		oldIm = newIm;
		//the actual iteration, the real and imaginary part are calculated
		newRe = oldRe * oldRe - oldIm * oldIm + pr;
		newIm = 2 * oldRe * oldIm + pi;
		//if the point is outside the circle with radius 2: stop
		if ((newRe * newRe + newIm * newIm) > 8)
			break;
	}
	//use color model conversion to get rainbow palette, make brightness black if maxIterations reached
	//color = HSVtoRGB(ColorHSV(i % 256, 255, 255 * (i < maxIterations)));
	//draw the pixel
	cuda.screen[dim_i] = 0;
	/*
	if (i < mandelbrot.maxIteration)
	{
		int color = 0;
		color = (color << 8) + i & 255;
		color = (color << 8) + 0;
		color = (color << 8) + 0;
		color = (color << 8) + 0xFF;
		cuda.screen[dim_i] = color;
	}
	else if (i % 2 == 0)*/
	if (i < mandelbrot.maxIteration)
		cuda.screen[dim_i] = 0xFFFFFFFF;
}

int
mandelbrot_call(t_data *data, t_cuda *cuda)
{
	static t_mandelbrot	mandelbrot = {{0}, 1, -0.5, 0, 300};

	if (data->esdl->en.in.key[SDL_SCANCODE_LEFT] == 1)
		mandelbrot.moveX -= 0.01 / mandelbrot.zoom * 10;
	if (data->esdl->en.in.key[SDL_SCANCODE_RIGHT] == 1)
		mandelbrot.moveX += 0.01 / mandelbrot.zoom * 10;
	if (data->esdl->en.in.key[SDL_SCANCODE_UP] == 1)
		mandelbrot.moveY -= 0.01 / mandelbrot.zoom * 10;
	if (data->esdl->en.in.key[SDL_SCANCODE_DOWN] == 1)
		mandelbrot.moveY += 0.01 / mandelbrot.zoom * 10;

	if (data->esdl->en.in.button[SDL_BUTTON_LEFT] == 1)
		mandelbrot.zoom += 0.01 * mandelbrot.zoom;
	if (data->esdl->en.in.button[SDL_BUTTON_RIGHT] == 1)
		mandelbrot.zoom -= 0.01 * mandelbrot.zoom;

	if (data->esdl->en.in.key[SDL_SCANCODE_KP_PLUS] == 1)
	{
		mandelbrot.maxIteration *= 1.1;
		printf("Max iterations = %d\n", mandelbrot.maxIteration);
	}
	if (data->esdl->en.in.key[SDL_SCANCODE_KP_MINUS] == 1 && mandelbrot.maxIteration > 300)
	{
		mandelbrot.maxIteration /= 1.1;
		printf("Max iterations = %d\n", mandelbrot.maxIteration);
	}


	mandelbrot_kernel<<<cuda->gridSize, cuda->blockSize>>>(*cuda, mandelbrot);
	return (0);
}

void
mandelbrot(t_data *data)
{
	do_cuda(data, &mandelbrot_call);
}