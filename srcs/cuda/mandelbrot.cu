#include "hip/hip_runtime.h"
extern "C"
{
#include <header.h>
}
#include <hip/hip_runtime.h>
#include <mandelbrot.h>

__global__ void
mandelbrot_kernel(t_cuda cuda, t_mandelbrot mandelbrot)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int dim_i = y * cuda.rx + x;
	if ((x >= cuda.rx) || (y >= cuda.ry))
		return ;

    double pr, pi;
    double newRe, newIm, oldRe, oldIm;

	pr = (x - cuda.rx / 2) / (0.5 * mandelbrot.zoom * cuda.rx) + mandelbrot.moveX;
	pi = (y - cuda.ry / 2) / (0.5 * mandelbrot.zoom * cuda.ry) + mandelbrot.moveY;
	newRe = newIm = oldRe = oldIm = 0;

	int i = 0;
	while (((newRe * newRe + newIm * newIm) < 4) && i < mandelbrot.maxIteration)
	{
	    oldRe = newRe;
	    oldIm = newIm;
	    newRe = oldRe * oldRe - oldIm * oldIm + pr;
	    newIm = 2 * oldRe * oldIm + pi;
	    i++;
	}

    if(i == mandelbrot.maxIteration)
        cuda.screen[dim_i] = 0x00000000;
    else
    {
        double z = sqrt(newRe * newRe + newIm * newIm);
        int brightness = 256. * log2(1.75 + i - log2(log2(z))) / log2(double(mandelbrot.maxIteration));
        cuda.screen[dim_i] = brightness << 24 | (i % 255) << 16 | 255 << 8 | 255;
    }
}

int
mandelbrot_call(t_data *data, t_cuda *cuda)
{
	static t_mandelbrot	mandelbrot = {1, -0.5, 0, 200, {0}};

	if (data->esdl->en.in.key[SDL_SCANCODE_LEFT] == 1)
		mandelbrot.moveX -= 0.01 / mandelbrot.zoom * 10;
	if (data->esdl->en.in.key[SDL_SCANCODE_RIGHT] == 1)
		mandelbrot.moveX += 0.01 / mandelbrot.zoom * 10;
	if (data->esdl->en.in.key[SDL_SCANCODE_UP] == 1)
		mandelbrot.moveY -= 0.01 / mandelbrot.zoom * 10;
	if (data->esdl->en.in.key[SDL_SCANCODE_DOWN] == 1)
		mandelbrot.moveY += 0.01 / mandelbrot.zoom * 10;

	if (data->esdl->en.in.button[SDL_BUTTON_LEFT] == 1)
		mandelbrot.zoom += 0.01 * mandelbrot.zoom;
	if (data->esdl->en.in.button[SDL_BUTTON_RIGHT] == 1)
		mandelbrot.zoom -= 0.01 * mandelbrot.zoom;

	if (data->esdl->en.in.key[SDL_SCANCODE_KP_PLUS] == 1)
	{
		mandelbrot.maxIteration *= 1.1;
		printf("Max iterations = %d\n", mandelbrot.maxIteration);
	}
	if (data->esdl->en.in.key[SDL_SCANCODE_KP_MINUS] == 1 && mandelbrot.maxIteration > mandelbrot.maxIteration)
	{
		mandelbrot.maxIteration /= 1.1;
		printf("Max iterations = %d\n", mandelbrot.maxIteration);
	}


	mandelbrot_kernel<<<cuda->gridSize, cuda->blockSize>>>(*cuda, mandelbrot);
	return (0);
}

void
mandelbrot(t_data *data)
{
	do_cuda(data, &mandelbrot_call);
}