#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   mandelbrot.cu                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: fgundlac <marvin@42.fr>                    +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/09/16 15:28:52 by fgundlac          #+#    #+#             */
/*   Updated: 2016/09/16 15:28:54 by fgundlac         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C"
{
#include <header.h>
}
#include <hip/hip_runtime.h>
#include <mandelbrot.h>

__device__ int
mandelbrot_color(double new_re, double new_im, int i, int max_iteration)
{
	double		z;
	int			brightness;

	//z = sqrt(new_re * new_re + new_im * new_im);
	(void)z;
	brightness = 256. * log2(1.75 + i - log2(log2((double)(max_iteration / 3)))) / log2((double)(max_iteration));
	return (brightness << 24 | (i % 255)  << 16 | brightness << 8 | 255);
}

__global__ void
mandelbrot_kernel(t_cuda cuda, t_mandelbrot mandelbrot)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int dim_i = y * cuda.rx + x;
	if ((x >= cuda.rx) || (y >= cuda.ry))
		return ;

	double		pr, pi;
	double		zx, zy;
	double		zx2, zy2;
	int			i;

	pr = mandelbrot.cx + (x - cuda.rx / 2) * mandelbrot.zoom + mandelbrot.movex;
	pi = mandelbrot.cy + (y - cuda.ry / 2) * mandelbrot.zoom + mandelbrot.movey;
	zx = 0;
	zy = 0;
	i = 0;
	while (i < mandelbrot.maxiteration)
	{
		zx2 = zx * zx;
		zy2 = zy * zy;
		zy = 2 * zx * zy + pi;
		zx = zx2 - zy2 + pr;
		if (zx2 + zy2 >= 4)
			break ;
		i++;
	}
	if (i == mandelbrot.maxiteration)
		cuda.screen[dim_i] = 0xFFFFFFFF;
	else
		cuda.screen[dim_i] = (int)(i * 255 / mandelbrot.maxiteration) << 24 | (i % 255)  << 16 | 255 << 8 | 255;
}

/*mandelbrot
		zx2 = zx * zx;
		zy2 = zy * zy;
		zy = 2 * zx * zy + pi;
		zx = zx2 - zy2 + pr;
		if (zx2 + zy2 >= 4)
			break ;
*/

/*
mandelbrot2
		zy2 = zy * zy;
		zx2 = zx * zx;
		zx = (zx2 * zx) - 3 * zx * zy2 + pr;
		zy = 3 * zx2 * zy - (zy2 * zy) + pi;
		if (zx2 + zy2 >= 4)
			break ;
*/

/*
mandelbrot4
		zy2 = zy * zy;
		zx2 = zx * zx;
		zx = (zx2 * zx2) - (6 * zx2 * zy2) + (zy2 * zy2) + pr;
		zy = (4 * (zx2 * zx) * zy) - (4 * zx * (zy2 * zy)) + pi;
		if (zx2 + zy2 >= 4)
			break ;
*/

void
mandelbrot_input(t_data *data, t_mandelbrot *mandelbrot)
{
	mandelbrot->oldcx = mandelbrot->cx;
	mandelbrot->oldcy = mandelbrot->cy;

	if (data->esdl->en.in.key[SDL_SCANCODE_LEFT] == 1)
		mandelbrot->movex -= 0.0001 / mandelbrot->zoom;
	if (data->esdl->en.in.key[SDL_SCANCODE_RIGHT] == 1)
		mandelbrot->movex += 0.0001 / mandelbrot->zoom;
	if (data->esdl->en.in.key[SDL_SCANCODE_UP] == 1)
		mandelbrot->movey -= 0.0001 / mandelbrot->zoom;
	if (data->esdl->en.in.key[SDL_SCANCODE_DOWN] == 1)
		mandelbrot->movey += 0.0001 / mandelbrot->zoom;
	if (data->esdl->en.in.button[SDL_BUTTON_LEFT] == 1)
	{
		mandelbrot->zoom = mandelbrot->zoom / 1.05;
		mandelbrot->cx = (mandelbrot->oldcx) + (mandelbrot->mx * 0.05) * mandelbrot->zoom;
		mandelbrot->cy = (mandelbrot->oldcy) + (mandelbrot->my * 0.05) * mandelbrot->zoom;
		mandelbrot->maxiteration *= 1.0025;
	}
	if (data->esdl->en.in.button[SDL_BUTTON_RIGHT] == 1)
	{
		mandelbrot->zoom = mandelbrot->zoom * 1.05;
		mandelbrot->cx = (mandelbrot->oldcx) + (mandelbrot->mx * 0.05) * mandelbrot->zoom;
		mandelbrot->cy = (mandelbrot->oldcy) + (mandelbrot->my * 0.05) * mandelbrot->zoom;
		mandelbrot->maxiteration *= 0.9975;
	}
	if (data->esdl->en.in.key[SDL_SCANCODE_KP_PLUS] == 1)
		mandelbrot->maxiteration *= 1.1;
	if (data->esdl->en.in.key[SDL_SCANCODE_KP_MINUS] == 1)
		mandelbrot->maxiteration *= 0.9;
}

int
mandelbrot_call(t_data *data, t_cuda *cuda)
{
	static t_mandelbrot	mandelbrot = {(2.5 / SDL_RY), 0, 0, 400, 0, 0, 0, 0, 0, 0};

	mandelbrot.mx = data->esdl->en.in.m_x - SDL_RX / 2;
	mandelbrot.my = data->esdl->en.in.m_y - SDL_RY / 2;
	mandelbrot_input(data, &mandelbrot);
	mandelbrot_kernel<<<cuda->gridsize, cuda->blocksize>>>(*cuda, mandelbrot);
	return (0);
}

void
mandelbrot(t_data *data)
{
	do_cuda(data, &mandelbrot_call);
}
