#include "hip/hip_runtime.h"
#include <stdio.h>

extern "C" {
#include <header.h>
}

__global__ void
my_kernel(int *a, int rx, int ry)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int dim_i = y * rx + x;
	if ((x >= rx) || (y >= ry))
		return ;

	//each iteration, it calculates: newz = oldz*oldz + p, where p is the current pixel, and oldz stars at the origin
	double pr, pi;           //real and imaginary part of the pixel p
	double newRe, newIm, oldRe, oldIm;   //real and imaginary parts of new and old z
	double zoom = 1, moveX = -0.5, moveY = 0; //you can change these to zoom and change position
	int maxIterations = 300;//after how much iterations the function should stop

	//calculate the initial real and imaginary part of z, based on the pixel location and zoom and position values
	pr = 1.5 * (x - rx / 2) / (0.5 * zoom * rx) + moveX;
	pi = (y - ry / 2) / (0.5 * zoom * ry) + moveY;
	newRe = newIm = oldRe = oldIm = 0; //these should start at 0,0

	int i;
	//start the iteration process
	for(i = 0; i < maxIterations; i++)
	{
		//remember value of previous iteration
		oldRe = newRe;
		oldIm = newIm;
		//the actual iteration, the real and imaginary part are calculated
		newRe = oldRe * oldRe - oldIm * oldIm + pr;
		newIm = 2 * oldRe * oldIm + pi;
		//if the point is outside the circle with radius 2: stop
		if ((newRe * newRe + newIm * newIm) > 4)
			break;
	}
	//use color model conversion to get rainbow palette, make brightness black if maxIterations reached
	//color = HSVtoRGB(ColorHSV(i % 256, 255, 255 * (i < maxIterations)));
	//draw the pixel
	if (i < maxIterations)
		a[dim_i] = 0xFFFFFFFF;
}

extern "C" void
mandelbrot(t_data *data, SDL_Surface *surf)
{
	static int *a_h = NULL;
	static int *a_d = NULL;  // Pointer to host & device arrays

	size_t size = SDL_RX * SDL_RY * sizeof(int);

	if (a_h == NULL)
		a_h = (int *)malloc(size);        // Allocate array on host

	if (a_d == NULL)
		hipMalloc((void **) &a_d, size);   // Allocate array on device

	// Initialize host array and copy it to CUDA device
	for (int i = 0; i < SDL_RX * SDL_RY; i++)
		a_h[i] = (int)0;
	
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);

	dim3 blockSize(32, 32);
	int bx = (SDL_RX + blockSize.x - 1) / blockSize.x;
	int by = (SDL_RY + blockSize.y - 1) / blockSize.y;
	dim3 gridSize = dim3(bx, by);

	my_kernel <<< gridSize, blockSize >>> (a_d, SDL_RX, SDL_RY);

	//hipDeviceSynchronize();

	// Retrieve result from device and store it in host array
	hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < SDL_RY; i++)
	{
		for (int j = 0; j < SDL_RX; j++)
		{
			Esdl_put_pixel(surf, j, i, a_h[i * SDL_RX + j]);
		}
	}

	//cleanup
	if (data->esdl->run == 0)
	{
		printf("QUIT BITCH !\n");
		free(a_h);
		hipFree(a_d);
	}

	//my_kernel<<<1,1>>>();
	//hipDeviceSynchronize();



	(void)data;
	(void)surf;
}