/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda.cu                                            :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: fgundlac <marvin@42.fr>                    +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/09/16 15:29:40 by fgundlac          #+#    #+#             */
/*   Updated: 2016/09/16 15:29:41 by fgundlac         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C"
{
#include <header.h>
}
#include <hip/hip_runtime.h>

//SOME CHECK TO DO MALLOC ETC...

void
do_cuda(t_data *data, int (*f)(t_data *, t_cuda *))
{
	static t_cuda cuda = {NULL};

	size_t size = SDL_RY * SDL_RX * data->surf->format->BytesPerPixel;

	if (cuda.screen == NULL)
	{
		hipMalloc((void **)&cuda.screen, size);
		cuda.blocksize = dim3(32, 32);
		cuda.bx = (SDL_RX + cuda.blocksize.x - 1) / cuda.blocksize.x;
		cuda.by = (SDL_RY + cuda.blocksize.y - 1) / cuda.blocksize.y;
		cuda.gridsize = dim3(cuda.bx, cuda.by);
		cuda.rx = SDL_RX;
		cuda.ry = SDL_RY;
	}

	f(data, &cuda);
	hipDeviceSynchronize();

	SDL_LockSurface(data->surf);
	hipMemcpy(data->surf->pixels, cuda.screen, size, hipMemcpyDeviceToHost);
	SDL_UnlockSurface(data->surf);

	if (data->esdl->run == 0)
		hipFree(cuda.screen);
}
