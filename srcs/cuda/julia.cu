#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   julia.cu                                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: fgundlac <marvin@42.fr>                    +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/09/16 15:28:59 by fgundlac          #+#    #+#             */
/*   Updated: 2016/09/16 15:29:00 by fgundlac         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C"
{
#include <header.h>
}
#include <hip/hip_runtime.h>
#include <julia.h>

__device__ int
julia_color(double new_re, double new_im, int i, int max_iteration)
{
	double			z;
	int				brightness;

	z = sqrt(new_re * new_re + new_im * new_im);
	brightness = 256. * log2(1.75 + i - log2(log2(z))) / log2((double)(max_iteration));
	return (brightness << 24 | (i % 255) << 16 | 255 << 8 | 255);
}

__global__ void
julia_kernel(t_cuda cuda, t_julia julia)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int dim_i = y * cuda.rx + x;
	if ((x >= cuda.rx) || (y >= cuda.ry))
		return ;

   	double			pr;
	double			pi;
	double			new_re;
	double			new_im;
	double			old_re;
	double			old_im;
	int				i;

	pr = 0.001 * julia.mx;
	pi = 0.001 * julia.my;
	new_re = (x - cuda.rx / 2) / (0.5 * julia.zoom * cuda.rx) + julia.movex;
	new_im = (y - cuda.ry / 2) / (0.5 * julia.zoom * cuda.ry) + julia.movey;
	i = 0;
	while (((new_re * new_re + new_im * new_im) < 4) && i < julia.maxiteration)
	{
		old_re = new_re;
		old_im = new_im;
		new_re = old_re * old_re - old_im * old_im + pr;
		new_im = 2 * old_re * old_im + pi;
		i++;
	}
	cuda.screen[dim_i] = julia_color(new_re, new_im, i, julia.maxiteration);
}

int
julia_call(t_data *data, t_cuda *cuda)
{
	static t_julia julia = {0, 0, 1, 0, 0, 300};
	julia.mx = data->esdl->en.in.m_x;
	julia.my = data->esdl->en.in.m_y;

	if (data->esdl->en.in.key[SDL_SCANCODE_LEFT] == 1)
		julia.movex -= 0.01 / julia.zoom * 10;
	if (data->esdl->en.in.key[SDL_SCANCODE_RIGHT] == 1)
		julia.movex += 0.01 / julia.zoom * 10;
	if (data->esdl->en.in.key[SDL_SCANCODE_UP] == 1)
		julia.movey -= 0.01 / julia.zoom * 10;
	if (data->esdl->en.in.key[SDL_SCANCODE_DOWN] == 1)
		julia.movey += 0.01 / julia.zoom * 10;

	if (data->esdl->en.in.button[SDL_BUTTON_LEFT] == 1)
		julia.zoom += 0.01 * julia.zoom;
	if (data->esdl->en.in.button[SDL_BUTTON_RIGHT] == 1)
		julia.zoom -= 0.01 * julia.zoom;

	if (data->esdl->en.in.key[SDL_SCANCODE_KP_PLUS] == 1)
	{
		julia.maxiteration *= 1.1;
		printf("Max iterations = %d\n", julia.maxiteration);
	}
	if (data->esdl->en.in.key[SDL_SCANCODE_KP_MINUS] == 1)
	{
		julia.maxiteration *= 0.9;
		printf("Max iterations = %d\n", julia.maxiteration);
	}

	julia_kernel<<<cuda->gridsize, cuda->blocksize>>>(*cuda, julia);
	return (0);
}

void
julia(t_data *data)
{
	do_cuda(data, &julia_call);
}
